#include "hip/hip_runtime.h"
/* Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "NvPipe.h"

#ifdef NVPIPE_WITH_ENCODER
#include "NvCodec/NvEncoder/NvEncoderCuda.h"
#endif

#ifdef NVPIPE_WITH_DECODER
#include "NvCodec/NvDecoder/NvDecoder.h"
#endif

#include "Utils/ColorSpace.h"
#include "Utils/NvCodecUtils.h"

#include <memory>
#include <iostream>
#include <string>
#include <sstream>
#include <unordered_map>
#include <mutex>
#include <queue>
#include <thread>
#include <atomic>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifdef NVPIPE_WITH_OPENGL
#include <cuda_gl_interop.h>
#endif

#ifdef _DEBUG
#define DEBUG_LOG(fmt, ...) (::fprintf(stderr, fmt, __VA_ARGS__))
#else
#define DEBUG_LOG(fmt, ...) 
#endif

class Exception
{
public:
	Exception(std::string msg) : message(msg) {}
	std::string getErrorString() const { return message; }
public:
	std::string message;
};

template<typename T>
static bool CheckInsideQueueRange(T head, T tail, T tocheck) {	//Check when on a circular buffer, given a range[tail, head), is tocheck inside the range.
	if (head >= tail) {
		return (tocheck < head && tocheck >= tail);	//Didn't check, hope this is right.
	}
	else {
		return (tocheck < head || tocheck >= tail);
	}
}

inline void CUDA_THROW(hipError_t code, std::string errorMessage)
{
	if (hipSuccess != code) {
		throw Exception(errorMessage + " (Error " + std::to_string(code) + ": " + std::string(hipGetErrorString(code)) + ")");
	}
}

inline bool isDevicePointer(const void* ptr)
{
	struct hipPointerAttribute_t attr;
	const hipError_t perr = hipPointerGetAttributes(&attr, ptr);

#if (CUDA_VERSION >= 10000)
	return (perr == hipSuccess) && (attr.type != hipMemoryTypeHost);
#else
	return (perr == hipSuccess) && (attr.memoryType != hipMemoryTypeHost);
#endif
}

inline uint64_t getFrameSize(NvPipe_Format format, uint32_t width, uint32_t height)
{
	if (format == NVPIPE_RGBA32)
		return width * height * 4;
	else if (format == NVPIPE_UINT4)
		return width * height / 2;
	else if (format == NVPIPE_UINT8)
		return width * height;
	else if (format == NVPIPE_UINT16)
		return width * height * 2;
	else if (format == NVPIPE_UINT32)
		return width * height * 4;

	return 0;
}


__global__
void uint4_to_nv12(const uint8_t* src, uint32_t srcPitch, uint8_t* dst, uint32_t dstPitch, uint32_t width, uint32_t height)
{
	// one thread per pixel
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		const uint32_t i = y * srcPitch + x / 2;
		const uint32_t j = y * dstPitch + x;

		// Extend 4 bit to 8 bits
		// Even thread: higher 4 bits, odd thread: lower 4 bits
		dst[j] = (x & 1 == 1) ? (src[i] & 0xF) : ((src[i] & 0xF0) >> 4);

		// Blank UV channel
		if (y < height / 2)
		{
			uint8_t* UV = dst + dstPitch * (height + y);
			UV[x] = 0;
		}
	}
}

__global__
void nv12_to_uint4(const uint8_t* src, uint32_t srcPitch, uint8_t* dst, uint32_t dstPitch, uint32_t width, uint32_t height)
{
	// one thread per TWO pixels
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (2 * x < width && y < height)
	{
		const uint32_t i = y * srcPitch + 2 * x;
		const uint32_t j = y * dstPitch + x;

		// Merge lower 4 bits of two Y bytes to one output byte
		uint8_t v = (src[i] & 0xF) << 4;

		if (2 * x + 1 < width)
			v = v | (src[i + 1] & 0xF);

		dst[j] = v;
	}
}

__global__
void uint8_to_nv12(const uint8_t* src, uint32_t srcPitch, uint8_t* dst, uint32_t dstPitch, uint32_t width, uint32_t height)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		const uint32_t i = y * srcPitch + x;
		const uint32_t j = y * dstPitch + x;

		// Copy grayscale image to Y channel
		dst[j] = src[i];

		// Blank UV channel
		if (y < height / 2)
		{
			uint8_t* UV = dst + dstPitch * (height + y);
			UV[x] = 0;
		}
	}
}

__global__
void nv12_to_uint8(const uint8_t* src, uint32_t srcPitch, uint8_t* dst, uint32_t dstPitch, uint32_t width, uint32_t height)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		const uint32_t i = y * srcPitch + x;
		const uint32_t j = y * dstPitch + x;

		// Copy Y channel to grayscale image
		dst[j] = src[i];

	}
}

__global__
void uint16_to_nv12(const uint8_t* src, uint32_t srcPitch, uint8_t* dst, uint32_t dstPitch, uint32_t width, uint32_t height)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		const uint32_t i = y * srcPitch + 2 * x;
		const uint32_t j = y * dstPitch + x;

		// Copy higher byte to left half of Y channel
		dst[j] = src[i];

		// Copy lower byte to right half of Y channel
		dst[j + width] = src[i + 1];

		// Blank UV channel
		if (y < height / 2)
		{
			uint8_t* UV = dst + dstPitch * (height + y);
			UV[2 * x + 0] = 0;
			UV[2 * x + 1] = 0;
		}
	}
}

__global__
void nv12_to_uint16(const uint8_t* src, uint32_t srcPitch, uint8_t* dst, uint32_t dstPitch, uint32_t width, uint32_t height)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		const uint32_t i = y * srcPitch + x;
		const uint32_t j = y * dstPitch + 2 * x;

		// Copy higher byte from left half of Y channel
		dst[j] = src[i];

		// Copy lower byte from right half of Y channel
		dst[j + 1] = src[i + width];
	}
}

__global__
void uint32_to_nv12(const uint8_t* src, uint32_t srcPitch, uint8_t* dst, uint32_t dstPitch, uint32_t width, uint32_t height)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		const uint32_t i = y * srcPitch + 4 * x;
		const uint32_t j = y * dstPitch + x;

		// Copy highest byte to left quarter of Y channel,
		// ...
		// Copy lowest byte to right quarter of Y channel
		dst[j] = src[i];
		dst[j + width] = src[i + 1];
		dst[j + 2 * width] = src[i + 2];
		dst[j + 3 * width] = src[i + 3];

		// Blank UV channel
		if (y < height / 2)
		{
			uint8_t* UV = dst + dstPitch * (height + y);
			UV[4 * x + 0] = 0;
			UV[4 * x + 1] = 0;
			UV[4 * x + 2] = 0;
			UV[4 * x + 3] = 0;
		}
	}
}

__global__
void nv12_to_uint32(const uint8_t* src, uint32_t srcPitch, uint8_t* dst, uint32_t dstPitch, uint32_t width, uint32_t height)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		const uint32_t i = y * srcPitch + x;
		const uint32_t j = y * dstPitch + 4 * x;

		// Copy highest byte from left quarter of Y channel
		// ...
		// Copy lowest byte from right quarter of Y channel
		dst[j] = src[i];
		dst[j + 1] = src[i + width];
		dst[j + 2] = src[i + 2 * width];
		dst[j + 3] = src[i + 3 * width];

	}
}

#ifdef NVPIPE_WITH_OPENGL
/**
 * @brief Utility class for managing CUDA-GL interop graphics resources.
 */
class GraphicsResourceRegistry
{
public:
	virtual ~GraphicsResourceRegistry()
	{
		// Unregister all
		for (auto& r : this->registeredPBOs)
			CUDA_THROW(hipGraphicsUnregisterResource(r.second.graphicsResource),
				"Failed to unregister PBO graphics resource");

		for (auto& r : this->registeredTextures) {
			auto result = hipGraphicsUnregisterResource(r.second.graphicsResource);
			//CUDA_THROW(result,"Failed to unregister texture graphics resource");  

			//This exception is suppressed in NvPipeUnity.
			//Unregister could throw due to early destroyed gl resource
			//In Unity plugin code, we can't hack into gl resource destruction. 
			//So when we press Stop button, all render textures are destroyed, and NvPipe is destructed, then an exception is thrown.
			//We can't handle the exception since it's hidden inside C interface.

			//So just disable it.
		}
	}

	hipGraphicsResource_t getTextureGraphicsResource(uint32_t texture, uint32_t target, uint32_t width, uint32_t height, uint32_t flags)
	{
		// Check if texture needs to be (re)registered
		RegisteredTexture& reg = this->registeredTextures[texture];

		if (reg.width != width || reg.height != height || reg.target != target) {
			if (reg.graphicsResource) {
				CUDA_THROW(hipGraphicsUnregisterResource(reg.graphicsResource),
					"Failed to unregister texture graphics resource");

				reg.graphicsResource = nullptr;
			}

			CUDA_THROW(hipGraphicsGLRegisterImage(&reg.graphicsResource, texture, target, flags),
				"Failed to register texture as graphics resource");

			reg.width = width;
			reg.height = height;
			reg.target = target;
		}

		return reg.graphicsResource;
	}

	hipGraphicsResource_t getPBOGraphicsResource(uint32_t pbo, uint32_t width, uint32_t height, uint32_t flags)
	{
		// Check if PBO needs to be (re)registered
		RegisteredPBO& reg = this->registeredPBOs[pbo];

		if (reg.width != width || reg.height != height) {
			if (reg.graphicsResource) {
				CUDA_THROW(hipGraphicsUnregisterResource(reg.graphicsResource),
					"Failed to unregister PBO graphics resource");

				reg.graphicsResource = nullptr;
			}

			CUDA_THROW(hipGraphicsGLRegisterBuffer(&reg.graphicsResource, pbo, flags),
				"Failed to register PBO as graphics resource");

			reg.width = width;
			reg.height = height;
		}

		return reg.graphicsResource;
	}

private:
	struct RegisteredTexture
	{
		hipGraphicsResource_t graphicsResource = nullptr;
		uint32_t width = 0;
		uint32_t height = 0;
		uint32_t target = 0;
	};
	std::unordered_map<uint32_t, RegisteredTexture> registeredTextures;

	struct RegisteredPBO
	{
		hipGraphicsResource_t graphicsResource = nullptr;
		uint32_t width = 0;
		uint32_t height = 0;
	};
	std::unordered_map<uint32_t, RegisteredPBO> registeredPBOs;
};
#endif


#ifdef NVPIPE_WITH_ENCODER

inline std::string EncErrorCodeToString(NVENCSTATUS code)
{
	std::vector<std::string> errors = {
		"NV_ENC_SUCCESS",
		"NV_ENC_ERR_NO_ENCODE_DEVICE",
		"NV_ENC_ERR_UNSUPPORTED_DEVICE",
		"NV_ENC_ERR_INVALID_ENCODERDEVICE",
		"NV_ENC_ERR_INVALID_DEVICE",
		"NV_ENC_ERR_DEVICE_NOT_EXIST",
		"NV_ENC_ERR_INVALID_PTR",
		"NV_ENC_ERR_INVALID_EVENT",
		"NV_ENC_ERR_INVALID_PARAM",
		"NV_ENC_ERR_INVALID_CALL",
		"NV_ENC_ERR_OUT_OF_MEMORY",
		"NV_ENC_ERR_ENCODER_NOT_INITIALIZED",
		"NV_ENC_ERR_UNSUPPORTED_PARAM",
		"NV_ENC_ERR_LOCK_BUSY",
		"NV_ENC_ERR_NOT_ENOUGH_BUFFER",
		"NV_ENC_ERR_INVALID_VERSION",
		"NV_ENC_ERR_MAP_FAILED",
		"NV_ENC_ERR_NEED_MORE_INPUT",
		"NV_ENC_ERR_ENCODER_BUSY",
		"NV_ENC_ERR_EVENT_NOT_REGISTERD",
		"NV_ENC_ERR_GENERIC",
		"NV_ENC_ERR_INCOMPATIBLE_CLIENT_KEY",
		"NV_ENC_ERR_UNIMPLEMENTED",
		"NV_ENC_ERR_RESOURCE_REGISTER_FAILED",
		"NV_ENC_ERR_RESOURCE_NOT_REGISTERED",
		"NV_ENC_ERR_RESOURCE_NOT_MAPPED"
	};

	if (code >= 0 && code < errors.size())
		return errors[code];

	return "Unknown error code";
}

/**
 * @brief Encoder implementation.
 */
class Encoder
{
public:
	Encoder(NvPipe_Format format, NvPipe_Codec codec, NvPipe_Compression compression, uint64_t bitrate, uint32_t targetFrameRate, uint32_t width, uint32_t height)
	{
		this->format = format;
		this->codec = codec;
		this->compression = compression;
		this->bitrate = bitrate;
		this->targetFrameRate = targetFrameRate;

		this->recreate(width, height);
	}

	virtual ~Encoder()
	{
		// Destroy encoder
		if (this->encoder)
		{
			std::vector<std::vector<uint8_t>> tmp;
			this->encoder->EndEncode(tmp);
			this->encoder->DestroyEncoder();
			this->encoder.reset();
		}

		// Free temporary device memory
		if (this->deviceBuffer)
			hipFree(this->deviceBuffer);
	}

	void setBitrate(uint64_t bitrate, uint32_t targetFrameRate)
	{
		NV_ENC_CONFIG config;
		memset(&config, 0, sizeof(config));
		config.version = NV_ENC_CONFIG_VER;
		config.rcParams.averageBitRate = bitrate;

		NV_ENC_RECONFIGURE_PARAMS reconfigureParams;
		memset(&reconfigureParams, 0, sizeof(reconfigureParams));
		reconfigureParams.version = NV_ENC_RECONFIGURE_PARAMS_VER;
		reconfigureParams.resetEncoder = 1;
		reconfigureParams.forceIDR = 1;
		reconfigureParams.reInitEncodeParams.encodeConfig = &config;

		encoder->GetInitializeParams(&reconfigureParams.reInitEncodeParams);
		reconfigureParams.reInitEncodeParams.frameRateNum = targetFrameRate;
		reconfigureParams.reInitEncodeParams.frameRateDen = 1;

		encoder->Reconfigure(&reconfigureParams);

		this->bitrate = bitrate;
		this->targetFrameRate = targetFrameRate;
	}

	uint64_t encode(const void* src, uint64_t srcPitch, uint8_t* dst, uint64_t dstSize, uint32_t width, uint32_t height, bool forceIFrame)
	{
		// Recreate encoder if size changed
		if (this->format == NVPIPE_UINT16)
			this->recreate(width * 2, height); // split into two adjecent tiles in Y channel
		else if (this->format == NVPIPE_UINT32)
			this->recreate(width * 4, height); // split into four adjecent tiles in Y channel
		else
			this->recreate(width, height);

		// RGBA can be directly copied from host or device
		if (this->format == NVPIPE_RGBA32)
		{
			const NvEncInputFrame* f = this->encoder->GetNextInputFrame();
			CUDA_THROW(hipMemcpy2D(f->inputPtr, f->pitch, src, srcPitch, width * 4, height, isDevicePointer(src) ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice),
				"Failed to copy input frame");
		}
		// Other formats need to be copied to the device and converted
		else
		{
			// Copy to device if necessary
			bool copyToDevice = !isDevicePointer(src);
			if (copyToDevice)
			{
				this->recreateDeviceBuffer(width, height);
				CUDA_THROW(hipMemcpy(this->deviceBuffer, src, getFrameSize(this->format, width, height), hipMemcpyHostToDevice),
					"Failed to copy input frame");
			}

			// Convert
			const NvEncInputFrame* f = this->encoder->GetNextInputFrame();

			if (this->format == NVPIPE_UINT4)
			{
				// one thread per pixel (extract 4 bit and copy to 8 bit)
				dim3 gridSize(width / 16 + 1, height / 2 + 1);
				dim3 blockSize(16, 2);

				uint4_to_nv12 << <gridSize, blockSize >> > ((uint8_t*)(copyToDevice ? this->deviceBuffer : src), srcPitch, (uint8_t*)f->inputPtr, f->pitch, width, height);
			}
			else if (this->format == NVPIPE_UINT8)
			{
				// one thread per pixel (copy 8 bit)
				dim3 gridSize(width / 16 + 1, height / 2 + 1);
				dim3 blockSize(16, 2);

				uint8_to_nv12 << <gridSize, blockSize >> > ((uint8_t*)(copyToDevice ? this->deviceBuffer : src), srcPitch, (uint8_t*)f->inputPtr, f->pitch, width, height);
			}
			else if (this->format == NVPIPE_UINT16)
			{
				// one thread per pixel (split 16 bit into 2x 8 bit)
				dim3 gridSize(width / 16 + 1, height / 2 + 1);
				dim3 blockSize(16, 2);

				uint16_to_nv12 << <gridSize, blockSize >> > ((uint8_t*)(copyToDevice ? this->deviceBuffer : src), srcPitch, (uint8_t*)f->inputPtr, f->pitch, width, height);
			}
			else if (this->format == NVPIPE_UINT32)
			{
				// one thread per pixel (split 32 bit into 4x 8 bit)
				dim3 gridSize(width / 16 + 1, height / 2 + 1);
				dim3 blockSize(16, 2);

				uint32_to_nv12 << <gridSize, blockSize >> > ((uint8_t*)(copyToDevice ? this->deviceBuffer : src), srcPitch, (uint8_t*)f->inputPtr, f->pitch, width, height);
			}
		}

		// Encode
		return this->encode(dst, dstSize, forceIFrame);
	}

#ifdef NVPIPE_WITH_OPENGL

	uint64_t encodeTexture(uint32_t texture, uint32_t target, uint8_t* dst, uint64_t dstSize, uint32_t width, uint32_t height, bool forceIFrame)
	{
		if (this->format != NVPIPE_RGBA32)
			throw Exception("The OpenGL interface only supports the RGBA32 format");

		// Recreate encoder if size changed
		this->recreate(width, height);

		// Map texture and copy input to encoder
		hipGraphicsResource_t resource = this->registry.getTextureGraphicsResource(texture, target, width, height, hipGraphicsRegisterFlagsReadOnly);
		CUDA_THROW(hipGraphicsMapResources(1, &resource),
			"Failed to map texture graphics resource");
		hipArray_t array;
		CUDA_THROW(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0),
			"Failed get texture graphics resource array");

		const NvEncInputFrame* f = this->encoder->GetNextInputFrame();
		CUDA_THROW(hipMemcpy2DFromArray(f->inputPtr, f->pitch, array, 0, 0, width * 4, height, hipMemcpyDeviceToDevice),
			"Failed to copy from texture array");

		// Encode
		uint64_t size = this->encode(dst, dstSize, forceIFrame);

		// Unmap texture
		CUDA_THROW(hipGraphicsUnmapResources(1, &resource),
			"Failed to unmap texture graphics resource");

		return size;
	}

	uint64_t encodePBO(uint32_t pbo, uint8_t* dst, uint64_t dstSize, uint32_t width, uint32_t height, bool forceIFrame)
	{
		if (this->format != NVPIPE_RGBA32)
			throw Exception("The OpenGL interface only supports the RGBA32 format");

		// Map PBO and copy input to encoder
		hipGraphicsResource_t resource = this->registry.getPBOGraphicsResource(pbo, width, height, hipGraphicsRegisterFlagsReadOnly);
		CUDA_THROW(hipGraphicsMapResources(1, &resource),
			"Failed to map PBO graphics resource");
		void* pboPointer;
		size_t pboSize;
		CUDA_THROW(hipGraphicsResourceGetMappedPointer(&pboPointer, &pboSize, resource),
			"Failed to get mapped PBO pointer");

		// Encode
		uint64_t size = this->encode(pboPointer, width * 4, dst, dstSize, width, height, forceIFrame);

		// Unmap PBO
		CUDA_THROW(hipGraphicsUnmapResources(1, &resource),
			"Failed to unmap PBO graphics resource");

		return size;
	}

#endif

protected:
	void recreate(uint32_t width, uint32_t height)
	{
		std::lock_guard<std::mutex> lock(Encoder::mutex);

		// Only recreate if necessary
		if (width == this->width && height == this->height)
			return;

		this->width = width;
		this->height = height;

		// Ensure we have a CUDA context
		CUDA_THROW(hipDeviceSynchronize(),
			"Failed to synchronize device");
		hipCtx_t cudaContext;
		hipCtxGetCurrent(&cudaContext);

		// Create encoder
		try
		{
			// Destroy previous encoder
			if (this->encoder)
			{
				std::vector<std::vector<uint8_t>> tmp;
				this->encoder->EndEncode(tmp);
				this->encoder->DestroyEncoder();
				this->encoder.reset();
			}

			NV_ENC_BUFFER_FORMAT bufferFormat = (this->format == NVPIPE_RGBA32) ? NV_ENC_BUFFER_FORMAT_ABGR : NV_ENC_BUFFER_FORMAT_NV12;
			this->encoder = std::unique_ptr<NvEncoderCuda>(new NvEncoderCuda(cudaContext, width, height, bufferFormat, 0));

			NV_ENC_INITIALIZE_PARAMS initializeParams = { NV_ENC_INITIALIZE_PARAMS_VER };
			NV_ENC_CONFIG encodeConfig = { NV_ENC_CONFIG_VER };
			initializeParams.encodeConfig = &encodeConfig;

			GUID codecGUID = (this->codec == NVPIPE_HEVC) ? NV_ENC_CODEC_HEVC_GUID : NV_ENC_CODEC_H264_GUID;

			GUID presetGUID = NV_ENC_PRESET_LOW_LATENCY_HQ_GUID;
			if (this->compression == NVPIPE_LOSSLESS)
				presetGUID = NV_ENC_PRESET_LOSSLESS_DEFAULT_GUID; // NV_ENC_PRESET_LOSSLESS_HP_GUID

			encoder->CreateDefaultEncoderParams(&initializeParams, codecGUID, presetGUID);

			initializeParams.encodeWidth = width;
			initializeParams.encodeHeight = height;
			initializeParams.frameRateNum = this->targetFrameRate;
			initializeParams.frameRateDen = 1;
			initializeParams.enablePTD = 1;

			encodeConfig.gopLength = NVENC_INFINITE_GOPLENGTH; // No B-frames
			encodeConfig.frameIntervalP = 1;

			if (this->codec == NVPIPE_H264)
				encodeConfig.encodeCodecConfig.h264Config.idrPeriod = NVENC_INFINITE_GOPLENGTH;
			else if (this->codec == NVPIPE_HEVC)
				encodeConfig.encodeCodecConfig.hevcConfig.idrPeriod = NVENC_INFINITE_GOPLENGTH;

			if (this->compression == NVPIPE_LOSSY)
			{
				encodeConfig.rcParams.averageBitRate = this->bitrate;
				encodeConfig.rcParams.rateControlMode = NV_ENC_PARAMS_RC_CBR_LOWDELAY_HQ;
				encodeConfig.rcParams.vbvBufferSize = encodeConfig.rcParams.averageBitRate * initializeParams.frameRateDen / initializeParams.frameRateNum; // bitrate / framerate = one frame
				encodeConfig.rcParams.maxBitRate = encodeConfig.rcParams.averageBitRate;
				encodeConfig.rcParams.vbvInitialDelay = encodeConfig.rcParams.vbvBufferSize;
			}

			encoder->CreateEncoder(&initializeParams);
		}
		catch (NVENCException & e)
		{
			throw Exception("Failed to create encoder (" + e.getErrorString() + ", error " + std::to_string(e.getErrorCode()) + " = " + EncErrorCodeToString(e.getErrorCode()) + ")");
		}
	}

	uint64_t encode(uint8_t* dst, uint64_t dstSize, bool forceIFrame)
	{
		std::vector<std::vector<uint8_t>> packets;

		try
		{
			if (forceIFrame)
			{
				NV_ENC_PIC_PARAMS params = {};
				params.encodePicFlags = NV_ENC_PIC_FLAG_FORCEIDR | NV_ENC_PIC_FLAG_OUTPUT_SPSPPS;

				this->encoder->EncodeFrame(packets, &params);
			}
			else
			{
				this->encoder->EncodeFrame(packets);
			}
		}
		catch (NVENCException & e)
		{
			throw Exception("Encode failed (" + e.getErrorString() + ", error " + std::to_string(e.getErrorCode()) + " = " + EncErrorCodeToString(e.getErrorCode()) + ")");
		}

		// Copy output
		uint64_t size = 0;
		for (auto& p : packets)
		{
			if (size + p.size() <= dstSize)
			{
				memcpy(dst + size, p.data(), p.size());
				size += p.size();
			}
			else
			{
				throw Exception("Encode output buffer overflow");
			}
		}

		return size;
	}

	void recreateDeviceBuffer(uint32_t width, uint32_t height)
	{
		// (Re)allocate temporary device memory if necessary
		uint64_t requiredSize = getFrameSize(this->format, width, height);

		if (this->deviceBufferSize < requiredSize)
		{
			if (this->deviceBuffer)
				hipFree(this->deviceBuffer);

			this->deviceBufferSize = requiredSize;
			CUDA_THROW(hipMalloc(&this->deviceBuffer, this->deviceBufferSize),
				"Failed to allocate temporary device memory");
		}
	}

protected:
	NvPipe_Format format;
	NvPipe_Codec codec;
	NvPipe_Compression compression;
	uint64_t bitrate;
	uint32_t targetFrameRate;
	uint32_t width = 0;
	uint32_t height = 0;

	std::unique_ptr<NvEncoderCuda> encoder;

	void* deviceBuffer = nullptr;
	uint64_t deviceBufferSize = 0;

	static std::mutex mutex;

#ifdef NVPIPE_WITH_OPENGL
	GraphicsResourceRegistry registry;
#endif
};

std::mutex Encoder::mutex;

#endif

#ifdef NVPIPE_WITH_DECODER

inline std::string DecErrorCodeToString(hipError_t code)
{
	const char* str = nullptr;
	hipDrvGetErrorName(code, &str);

	if (str)
		return std::string(str);

	return "Unknown error code";
}

/**
 * @brief Decoder implementation.
 */
class Decoder
{
public:
	Decoder(NvPipe_Format format, NvPipe_Codec codec, uint32_t width, uint32_t height)
	{
		this->format = format;
		this->codec = codec;

		this->recreate(width, height);
	}

	~Decoder()
	{
		// Free temporary device memory
		if (this->deviceBuffer)
			hipFree(this->deviceBuffer);
	}

	uint64_t decode(const uint8_t* src, uint64_t srcSize, void* dst, uint32_t width, uint32_t height)
	{
		// Recreate decoder if size changed
		if (this->format == NVPIPE_UINT16)
			this->recreate(width * 2, height); // split into two adjecent tiles in Y channel
		else if (this->format == NVPIPE_UINT32)
			this->recreate(width * 4, height); // split into four adjecent tiles in Y channel
		else
			this->recreate(width, height);

		// Decode
		uint8_t* decoded = this->decode(src, srcSize);

		if (nullptr != decoded)
		{
			// Allocate temporary device buffer if we need to copy to the host eventually
			bool copyToHost = !isDevicePointer(dst);
			if (copyToHost)
				this->recreateDeviceBuffer(width, height);

			// Convert to output format
			uint8_t* dstDevice = (uint8_t*)(copyToHost ? this->deviceBuffer : dst);

			if (this->format == NVPIPE_RGBA32)
			{
				Nv12ToColor32<RGBA32>(decoded, width, dstDevice, width * 4, width, height);
			}
			else if (this->format == NVPIPE_UINT4)
			{
				// one thread per TWO pixels (merge 2x4 bit to one byte per thread)
				dim3 gridSize(width / 16 / 2 + 1, height / 2 + 1);
				dim3 blockSize(16, 2);

				nv12_to_uint4 << <gridSize, blockSize >> > (decoded, this->decoder->GetDeviceFramePitch(), dstDevice, width / 2, width, height);
			}
			else if (this->format == NVPIPE_UINT8)
			{
				// one thread per pixel (copy 8 bit)
				dim3 gridSize(width / 16 + 1, height / 2 + 1);
				dim3 blockSize(16, 2);

				nv12_to_uint8 << <gridSize, blockSize >> > (decoded, this->decoder->GetDeviceFramePitch(), dstDevice, width, width, height);
			}
			else if (this->format == NVPIPE_UINT16)
			{
				// one thread per pixel (merge 2x8 bit into 16 bit pixels)
				dim3 gridSize(width / 16 + 1, height / 2 + 1);
				dim3 blockSize(16, 2);

				nv12_to_uint16 << <gridSize, blockSize >> > (decoded, this->decoder->GetDeviceFramePitch(), dstDevice, width * 2, width, height);
			}
			else if (this->format == NVPIPE_UINT32)
			{
				// one thread per pixel (merge 4x8 bit into 32 bit pixels)
				dim3 gridSize(width / 16 + 1, height / 2 + 1);
				dim3 blockSize(16, 2);

				nv12_to_uint32 << <gridSize, blockSize >> > (decoded, this->decoder->GetDeviceFramePitch(), dstDevice, width * 4, width, height);
			}

			// Copy to host if necessary
			if (copyToHost)
				CUDA_THROW(hipMemcpy(dst, this->deviceBuffer, getFrameSize(this->format, width, height), hipMemcpyDeviceToHost),
					"Failed to copy output to host memory");

			return getFrameSize(this->format, width, height);
		}

		return 0;
	}

#ifdef NVPIPE_WITH_OPENGL

	uint64_t decodeTexture(const uint8_t* src, uint64_t srcSize, uint32_t texture, uint32_t target, uint32_t width, uint32_t height)
	{
		if (this->format != NVPIPE_RGBA32)
			throw Exception("The OpenGL interface only supports the RGBA32 format");

		// Recreate decoder if size changed
		this->recreate(width, height);

		// Decode
		uint8_t* decoded = this->decode(src, srcSize);

		if (nullptr != decoded)
		{
			// Convert to RGBA
			this->recreateDeviceBuffer(width, height);
			Nv12ToColor32<RGBA32>(decoded, width, (uint8_t*)this->deviceBuffer, width * 4, width, height);

			// Copy output to texture
			hipGraphicsResource_t resource = this->registry.getTextureGraphicsResource(texture, target, width, height, hipGraphicsRegisterFlagsWriteDiscard);
			CUDA_THROW(hipGraphicsMapResources(1, &resource),
				"Failed to map texture graphics resource");
			hipArray_t array;
			CUDA_THROW(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0),
				"Failed get texture graphics resource array");
			CUDA_THROW(hipMemcpy2DToArray(array, 0, 0, this->deviceBuffer, width * 4, width * 4, height, hipMemcpyDeviceToDevice),
				"Failed to copy to texture array");
			CUDA_THROW(hipGraphicsUnmapResources(1, &resource),
				"Failed to unmap texture graphics resource");

			return width * height * 4;
		}

		return 0;
	}

	uint64_t decodePBO(const uint8_t* src, uint64_t srcSize, uint32_t pbo, uint32_t width, uint32_t height)
	{
		if (this->format != NVPIPE_RGBA32)
			throw Exception("The OpenGL interface only supports the RGBA32 format");

		// Map PBO for output
		hipGraphicsResource_t resource = this->registry.getPBOGraphicsResource(pbo, width, height, hipGraphicsRegisterFlagsWriteDiscard);
		CUDA_THROW(hipGraphicsMapResources(1, &resource),
			"Failed to map PBO graphics resource");
		void* pboPointer;
		size_t pboSize;
		CUDA_THROW(hipGraphicsResourceGetMappedPointer(&pboPointer, &pboSize, resource),
			"Failed to get mapped PBO pointer");

		// Decode
		uint64_t size = this->decode(src, srcSize, pboPointer, width, height);

		// Unmap PBO
		CUDA_THROW(hipGraphicsUnmapResources(1, &resource),
			"Failed to unmap PBO graphics resource");

		return size;
	}

#endif

private:
	void recreate(uint32_t width, uint32_t height)
	{
		std::lock_guard<std::mutex> lock(Decoder::mutex);

		// Only recreate if necessary
		if (width == this->width && height == this->height)
			return;

		this->width = width;
		this->height = height;

		// Ensure we have a CUDA context
		CUDA_THROW(hipDeviceSynchronize(),
			"Failed to synchronize device");
		hipCtx_t cudaContext;
		hipCtxGetCurrent(&cudaContext);

		// Create decoder
		try
		{
			// Destroy previous decoder
			if (this->decoder)
			{
				this->decoder.reset();
			}

			this->decoder = std::unique_ptr<NvDecoder>(new NvDecoder(cudaContext, width, height, true, (this->codec == NVPIPE_HEVC) ? cudaVideoCodec_HEVC : cudaVideoCodec_H264,/* &Decoder::mutex*/ nullptr, true));
		}
		catch (NVDECException & e)
		{
			throw Exception("Failed to create decoder (" + e.getErrorString() + ", error " + std::to_string(e.getErrorCode()) + " = " + DecErrorCodeToString(e.getErrorCode()) + ")");
		}
	}

	uint8_t* decode(const uint8_t* src, uint64_t srcSize)
	{
		int numFramesDecoded = 0;
		uint8_t** decodedFrames;
		int64_t* timeStamps;

		try
		{
			// Some cuvid implementations have one frame latency. Refeed frame into pipeline in this case.
			const uint32_t DECODE_TRIES = 3;
			for (uint32_t i = 0; (i < DECODE_TRIES) && (numFramesDecoded <= 0); ++i)
				this->decoder->Decode(src, srcSize, &decodedFrames, &numFramesDecoded, CUVID_PKT_ENDOFPICTURE, &timeStamps, this->n++);
		}
		catch (NVDECException & e)
		{
			throw Exception("Decode failed (" + e.getErrorString() + ", error " + std::to_string(e.getErrorCode()) + " = " + DecErrorCodeToString(e.getErrorCode()) + ")");
		}

		if (numFramesDecoded <= 0)
		{
			throw Exception("No frame decoded (Decoder expects encoded bitstream for a single complete frame. Accumulating partial data or combining multiple frames is not supported.)");
		}

		return decodedFrames[numFramesDecoded - 1];
	}

	void recreateDeviceBuffer(uint32_t width, uint32_t height)
	{
		// (Re)allocate temporary device memory if necessary
		uint64_t requiredSize = getFrameSize(this->format, width, height);

		if (this->deviceBufferSize < requiredSize)
		{
			if (this->deviceBuffer)
				hipFree(this->deviceBuffer);

			this->deviceBufferSize = requiredSize;
			CUDA_THROW(hipMalloc(&this->deviceBuffer, this->deviceBufferSize),
				"Failed to allocate temporary device memory");
		}
	}

private:
	NvPipe_Format format;
	NvPipe_Codec codec;
	uint32_t width = 0;
	uint32_t height = 0;

	std::unique_ptr<NvDecoder> decoder;
	int64_t n = 0;

	void* deviceBuffer = nullptr;
	uint64_t deviceBufferSize = 0;

	static std::mutex mutex;

#ifdef NVPIPE_WITH_OPENGL
	GraphicsResourceRegistry registry;
#endif
};

std::mutex Decoder::mutex;

#endif




/*
MIT License

Copyright (c) 2020 yangrc1234

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

*/

#ifdef NVPIPE_WITH_ENCODER
#ifdef NVPIPE_WITH_OPENGL
class AsyncTextureEncoder : public Encoder
{
	struct IntermediateBuffer {
		hipDeviceptr_t ptr;
		size_t pitch;
		~IntermediateBuffer()
		{
			hipFree(ptr);
		}
	};
public:
	static constexpr int kEncodeBufferCount = 3;
	AsyncTextureEncoder(NvPipe_Format format, NvPipe_Codec codec, NvPipe_Compression compression, uint64_t bitrate, uint32_t targetFrameRate, uint32_t width, uint32_t height) :
		Encoder(format, codec, compression, bitrate, targetFrameRate, width, height),
		m_clearedPtr(0), m_encodedPtr(0), m_pendingTaskPtr(0)
	{
		m_outputBufferSize = width * height * 4;
		for (size_t i = 0; i < 3; i++)
		{
			hipMemAllocPitch(&m_intermdiateBuffer[i].ptr, &m_intermdiateBuffer[i].pitch, width * 4, height, 16);
			this->m_outputBuffer[i] = std::make_unique<uint8_t[]>(m_outputBufferSize);
		}
		m_closed = false;
		m_encodeThread = std::make_unique<std::thread>(&AsyncTextureEncoder::encodeThread, this);
	}

	virtual ~AsyncTextureEncoder()
	{
		m_closed = true;
		m_encodeThread->join();
	}

	int encodeTextureAsync(uint32_t texture, uint32_t target, uint32_t width, uint32_t height, bool forceIFrame) {
		if (this->format != NVPIPE_RGBA32)
			throw Exception("The OpenGL interface only supports the RGBA32 format");

		if ((this->m_pendingTaskPtr + 1) % kEncodeBufferCount == this->m_clearedPtr)  //Encode/Clear task is too slow.
		{
			throw Exception("Encoder is too slow or task is not cleared, failed to enqueue new encode task. \n");
		}

		auto currentTaskIndex = this->m_pendingTaskPtr.load();

		// Map texture and copy input to encoder
		hipGraphicsResource_t resource = this->registry.getTextureGraphicsResource(texture, target, width, height, hipGraphicsRegisterFlagsReadOnly);
		CUDA_THROW(hipGraphicsMapResources(1, &resource),
			"Failed to map texture graphics resource");
		hipArray_t array;
		CUDA_THROW(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0),
			"Failed get texture graphics resource array");

		//Copy to intermediate buffer.
		CUDA_THROW(hipMemcpy2DFromArray(
			(void*)m_intermdiateBuffer[currentTaskIndex].ptr,
			m_intermdiateBuffer[currentTaskIndex].pitch,
			array,
			0, 0, width * 4, height, hipMemcpyDeviceToDevice),
			"Failed to copy memory to intermediate buffer."
		);

		// Unmap texture
		CUDA_THROW(hipGraphicsUnmapResources(1, &resource),
			"Failed to unmap texture graphics resource");

		AsyncTask task;
		task.forceIFrame = forceIFrame;
		task.height = height;
		task.width = width;
		task.isError = false;
		m_tasks[currentTaskIndex] = std::move(task);
		DEBUG_LOG("Encoder: %d task is in async queue now\n", currentTaskIndex);

		//Task is published, move pointer to next, and encode thread can operate on current task.
		//Atomic guarantees this happens after m_tasks[currentTaskIndex] = std::move(task);
		this->m_pendingTaskPtr = (this->m_pendingTaskPtr + 1) % kEncodeBufferCount;
		return currentTaskIndex;    //Return task number.
	}

	void encodeThread() {
		while (!m_closed)
		{
			if (m_encodedPtr == m_pendingTaskPtr)
			{
				std::this_thread::yield();
				continue;
			}
			auto& currTask = m_tasks[m_encodedPtr];

			DEBUG_LOG("Encoder thread: Encoding task: %d\n", m_encodedPtr.load());

			try
			{
				// Encode
				const NvEncInputFrame* f = this->encoder->GetNextInputFrame();
				CUDA_THROW(hipMemcpy2D(f->inputPtr, f->pitch,
					(void*)m_intermdiateBuffer[m_encodedPtr].ptr,
					m_intermdiateBuffer[m_encodedPtr].pitch,
					width * 4, currTask.height, hipMemcpyDeviceToDevice),
					"Failed to copy from texture array");
				uint64_t size = this->encode(m_outputBuffer[m_encodedPtr].get(), m_outputBufferSize, currTask.forceIFrame);
				m_tasks[m_encodedPtr].isError = false;
				m_tasks[m_encodedPtr].encodedSize = size;
			}
			catch (const Exception & e)
			{
				m_tasks[m_encodedPtr].isError = true;
				m_tasks[m_encodedPtr].error = e.message;
			}

			DEBUG_LOG("Encoding finished\n");
			m_encodedPtr = (m_encodedPtr + 1) % kEncodeBufferCount;
		}
	}

	void QueryTask(int taskIndex, bool* isDone, bool* isError, std::string* error) {
		DEBUG_LOG("Encoder Query: query %d\n", taskIndex);
		if (!CheckInsideQueueRange(m_pendingTaskPtr.load(), m_clearedPtr.load(), taskIndex)) {
			//This task doesn't eixsts.
			DEBUG_LOG("Encoder Query: task doesn't exists\n");
			throw Exception("Task doesn't exists");
		}
		if (CheckInsideQueueRange(m_pendingTaskPtr.load(), m_encodedPtr.load(), taskIndex)) {
			//Not encoded yet
			DEBUG_LOG("Encoder Query: task %d not encoded yet\n", taskIndex);
			*isDone = false;
			*isError = false;
			return;
		}
		if (CheckInsideQueueRange(m_encodedPtr.load(), m_clearedPtr.load(), taskIndex)) {
			//Done
			DEBUG_LOG("Encoder Query: task %d is done.\n", taskIndex);
			auto& task = m_tasks[taskIndex];
			*isDone = true;
			*isError = task.isError;
			if (task.isError)
				*error = task.error;
			return;
		}
		DEBUG_LOG("Query Task Exception Encoutnered, Current Status: Pending %d Encoding %d Clearing %d Query %d\n", m_pendingTaskPtr.load(), m_encodedPtr.load(), m_clearedPtr.load(), taskIndex);
		throw Exception("Unknown error");
	}

	uint8_t* AcquireTaskData(int taskIndex, uint64_t* encodeSize) {
		auto& task = m_tasks[taskIndex];
		if (!CheckInsideQueueRange(m_encodedPtr.load(), m_clearedPtr.load(), taskIndex)) {
			throw Exception("The task is not done yet!");
		}
		if (!task.isError)
			*encodeSize = m_tasks[taskIndex].encodedSize;
		return m_outputBuffer[taskIndex].get();
	}

	void ClearTask(int taskIndex) {
		if (taskIndex != m_clearedPtr) {
			throw Exception("Only next task could be cleared!");
		}
		if (taskIndex == m_encodedPtr) {
			throw Exception("The task is not finished yet!");
		}
		m_tasks[m_clearedPtr] = AsyncTask();
		m_clearedPtr = (1 + m_clearedPtr) % kEncodeBufferCount;
		DEBUG_LOG("Encoder: %d task is cleared\n", m_clearedPtr.load());
	}

private:

	struct AsyncTask
	{
		AsyncTask()
		{
		}
		uint32_t width;
		uint32_t height;
		bool forceIFrame;

		//bool isDone;	//isDone could be inferred from circular buffer pointers.
		bool isError;
		uint64_t encodedSize;
		std::string error;
	};
	std::atomic<bool> m_closed;
	std::atomic<int> m_pendingTaskPtr;
	std::atomic<int> m_encodedPtr;
	std::atomic<int> m_clearedPtr;
	IntermediateBuffer m_intermdiateBuffer[kEncodeBufferCount];
	std::unique_ptr<uint8_t[]> m_outputBuffer[kEncodeBufferCount];
	uint64_t m_outputBufferSize;
	AsyncTask m_tasks[kEncodeBufferCount];
	std::unique_ptr<std::thread> m_encodeThread;
};
#endif
#endif

struct Instance
{
#ifdef NVPIPE_WITH_ENCODER
	std::unique_ptr<Encoder> encoder;
#ifdef NVPIPE_WITH_OPENGL
	std::unique_ptr<AsyncTextureEncoder> asyncTextureEncoder;
#endif
#endif


#ifdef NVPIPE_WITH_DECODER
	std::unique_ptr<Decoder> decoder;
#endif


	std::string error;
};

std::string sharedError; // shared error code for create functions (NOT threadsafe)
std::unordered_map<uint32_t, std::shared_ptr<Instance>> g_pipes;
std::mutex g_pipeDictMutex;
uint32_t g_pipeCreationIndex = 1;	//Start with 1, since GetError requries a special value(0 here) for global error.

static void DeletePipe(uint32_t id) {
	std::lock_guard<std::mutex> lock(g_pipeDictMutex);
	g_pipes.erase(id);
}

static std::shared_ptr<Instance> GetPipe(uint32_t id) {
	std::lock_guard<std::mutex> lock(g_pipeDictMutex);
	auto ite = g_pipes.find(id);
	if (ite == g_pipes.end()) {
		return nullptr;
	}
	return ite->second;
}

static uint32_t InsertNewPipe(std::shared_ptr<Instance> instance) {
	std::lock_guard<std::mutex> lock(g_pipeDictMutex);
	auto index = g_pipeCreationIndex++;
	g_pipes[index] = instance;
	return index;
}

#ifdef NVPIPE_WITH_ENCODER

UNITY_INTERFACE_EXPORT uint32_t UNITY_INTERFACE_API NvPipe_CreateEncoder(NvPipe_Format format, NvPipe_Codec codec, NvPipe_Compression compression, uint64_t bitrate, uint32_t targetFrameRate, uint32_t width, uint32_t height)
{
	auto instance = std::make_shared<Instance>();

	try
	{
		instance->encoder = std::unique_ptr<Encoder>(new Encoder(format, codec, compression, bitrate, targetFrameRate, width, height));
		return InsertNewPipe(instance);
	}
	catch (Exception & e)
	{
		sharedError = e.getErrorString();
		return 0;
	}

	return 0;
}

#ifdef NVPIPE_WITH_OPENGL
UNITY_INTERFACE_EXPORT uint32_t UNITY_INTERFACE_API NvPipe_CreateTextureAsyncEncoder(NvPipe_Format format, NvPipe_Codec codec, NvPipe_Compression compression, uint64_t bitrate, uint32_t targetFrameRate, uint32_t width, uint32_t height)
{
	auto instance = std::make_shared<Instance>();

	try
	{
		instance->asyncTextureEncoder = std::make_unique<AsyncTextureEncoder>(format, codec, compression, bitrate, targetFrameRate, width, height);
		return InsertNewPipe(instance);
	}
	catch (Exception & e)
	{
		sharedError = e.getErrorString();
		return 0;
	}

	return 0;
}
#endif

UNITY_INTERFACE_EXPORT void UNITY_INTERFACE_API NvPipe_SetBitrate(uint64_t pipe, uint64_t bitrate, uint32_t targetFrameRate)
{
	auto instance = GetPipe(pipe);
	if (instance == nullptr)
		return;

	if (!instance->encoder)
	{
		instance->error = "Invalid NvPipe encoder.";
		return;
	}

	try
	{
		return instance->encoder->setBitrate(bitrate, targetFrameRate);
	}
	catch (Exception & e)
	{
		instance->error = e.getErrorString();
	}
}

UNITY_INTERFACE_EXPORT uint64_t UNITY_INTERFACE_API NvPipe_Encode(uint32_t pipe, const void* src, uint64_t srcPitch, uint8_t* dst, uint64_t dstSize, uint32_t width, uint32_t height, bool forceIFrame)
{
	auto instance = GetPipe(pipe);
	if (instance == nullptr)
		return 0;
	if (!instance->encoder)
	{
		instance->error = "Invalid NvPipe encoder.";
		return 0;
	}

	try
	{
		return instance->encoder->encode(src, srcPitch, dst, dstSize, width, height, forceIFrame);
	}
	catch (Exception & e)
	{
		instance->error = e.getErrorString();
		return 0;
	}
}

#ifdef NVPIPE_WITH_OPENGL

UNITY_INTERFACE_EXPORT uint64_t UNITY_INTERFACE_API NvPipe_EncodeTexture(uint32_t pipe, uint32_t texture, uint32_t target, uint8_t* dst, uint64_t dstSize, uint32_t width, uint32_t height, bool forceIFrame)
{
	auto instance = GetPipe(pipe);

	if (instance == nullptr)
		return 0;

	if (!instance->encoder)
	{
		instance->error = "Invalid NvPipe encoder.";
		return 0;
	}

	try
	{
		return instance->encoder->encodeTexture(texture, target, dst, dstSize, width, height, forceIFrame);
	}
	catch (Exception & e)
	{
		instance->error = e.getErrorString();
		return 0;
	}
}


/*
==================================
Async OpenGL Texture Encoding.
==================================
*/
struct MainThreadPendingTask {	//Tasks from main thread.
	MainThreadPendingTask() = default;
	MainThreadPendingTask(std::shared_ptr<Instance> nvp, uint32_t texture, uint32_t width, uint32_t height, bool forceIFrame)
	{
		this->pipe = nvp;
		this->texture = texture;
		this->width = width;
		this->height = height;
		this->forceIFrame = forceIFrame;

		this->isDone = false;
		this->isError = false;
	}

	std::shared_ptr<Instance> pipe;
	uint32_t texture;
	uint32_t width;
	uint32_t height;
	bool forceIFrame;
	int mainThreadTaskIndex;

	//Available once the renderthread submitted task to encoder
	int encoderTaskIndex;

	//Available once results are polled from encoder.
	bool isDone;
	bool isError;
	uint8_t* resultBuffer;
	std::string error;
	uint64_t encodedSize;
};

static constexpr int MAX_PENDING_TASK_COUNT = 20;	//Up to 20 tasks could exist at the same time.
static MainThreadPendingTask mainThreadPendingTasks[MAX_PENDING_TASK_COUNT];	//This is a circular buffer.

static std::atomic<uint32_t> g_pendingTaskPtr(0);		//circular buffer pointer.
static std::atomic<uint32_t> g_submittedTaskPtr(0);		//circular buffer pointer.
static std::atomic<uint32_t> g_doneTaskPtr(0);
static std::atomic<uint32_t> g_cleardTaskPtr(0);

static std::mutex g_destructMutex;	/*Used when resetting encode tasks. Reset only callde by main thread, so only RenderThread calls will need this mutex.*/

/*
Called in main thread, to clear all async encoding tasks.
But actual encoders and tasks inside it won't be destructed.
*/
UNITY_INTERFACE_EXPORT void UNITY_INTERFACE_API NvPipe_ResetEncodeTasks() {
	std::lock_guard<std::mutex> lock(g_destructMutex);
	for (size_t i = 0; i < MAX_PENDING_TASK_COUNT; i++)
	{
		mainThreadPendingTasks[i] = MainThreadPendingTask();
	}
	g_pendingTaskPtr = 0;
	g_submittedTaskPtr = 0;
	g_doneTaskPtr = 0;
	g_cleardTaskPtr = 0;

	DEBUG_LOG("async encode queue reset\n");
}

/*Called in main thread, to enqueue a new task.*/
UNITY_INTERFACE_EXPORT uint32_t UNITY_INTERFACE_API NvPipe_QueueEncodeTaskInMainThread(uint32_t nvp, uint32_t texture, uint32_t width, uint32_t height, bool forceIFrame) {
	auto pipe = GetPipe(nvp);
	if (pipe == nullptr)
		return 0;
	if ((g_pendingTaskPtr + 1) % MAX_PENDING_TASK_COUNT == g_cleardTaskPtr) {	//Reached maximum submit tasks per frame, or earlier tasks are not cleared yet.
		static char msgBuffer[200];
		sprintf(msgBuffer, "Maximum task count reached. Did you forget to clear task, or submitted too many tasks(%d) at once?", MAX_PENDING_TASK_COUNT);
		pipe->error = msgBuffer;
		return 0;
	}

	if (pipe->asyncTextureEncoder == nullptr) {
		pipe->error = "Invalid async texture encoder";
		return 0;
	}

	auto ptr = g_pendingTaskPtr.load();
	mainThreadPendingTasks[g_pendingTaskPtr] = MainThreadPendingTask(pipe, texture, width, height, forceIFrame);
	g_pendingTaskPtr = (1 + g_pendingTaskPtr) % MAX_PENDING_TASK_COUNT;
	DEBUG_LOG("async encode task enqueued, task index %d\n", ptr);
	return ptr;
}

/*
Called by render thread
To move all pending tasks into corresponding encoders,
and update all task infos from encoder.
*/
UNITY_INTERFACE_EXPORT void UNITY_INTERFACE_API NvPipe_RenderThreadPoll(int)
{
	std::lock_guard<std::mutex> lock(g_destructMutex);
	DEBUG_LOG("RTP: Render thread polling(Move to encoder)\n");
	while (g_submittedTaskPtr != g_pendingTaskPtr)
	{
		DEBUG_LOG("RTP: %d, %d\n", g_submittedTaskPtr.load(), g_pendingTaskPtr.load());
		auto& task = mainThreadPendingTasks[g_submittedTaskPtr];
		//Allocate buffer

		try     //Error here is not stored in instance.error, since we're in render thread. Put error along inside SubmittedTask.
		{
			task.encoderTaskIndex = task.pipe->asyncTextureEncoder->encodeTextureAsync(task.texture, GL_TEXTURE_2D, task.width, task.height, task.forceIFrame);

			DEBUG_LOG("RTP: %d entered encoder queue\n", g_submittedTaskPtr.load());
			task.isDone = false;
			task.isError = false;
		}
		catch (const Exception & e)
		{
			task.isDone = true;
			task.isError = true;
			task.error = e.message;
			DEBUG_LOG("RTP: %d failed to enqueue to encoder, error:%s\n", g_submittedTaskPtr.load(), e.message.c_str());
		}
		g_submittedTaskPtr = (g_submittedTaskPtr + 1) % MAX_PENDING_TASK_COUNT;
	}

	DEBUG_LOG("RTP: Render thread polling(Check done)\n");
	//Query any task is done.
	while (g_doneTaskPtr != g_submittedTaskPtr)
	{
		DEBUG_LOG("RTP: %d, %d\n", g_doneTaskPtr.load(), g_submittedTaskPtr.load());
		auto& task = mainThreadPendingTasks[g_doneTaskPtr];

		if (!task.isDone)	//Try to get task done.
		{
			try
			{	//Query task status from encoder.
				task.pipe->asyncTextureEncoder->QueryTask(task.encoderTaskIndex,
					&task.isDone,
					&task.isError,
					&task.error);

				if (task.isDone)
					DEBUG_LOG("RTP: Task set to done\n");
				if (task.isError)
					DEBUG_LOG("RTP: Task done with error: %s\n", task.error.c_str());

				if (task.isDone) {
					if (!task.isError) {
						//Get buffer back.
						task.resultBuffer = task.pipe->asyncTextureEncoder->AcquireTaskData(task.encoderTaskIndex,
							&task.encodedSize);
					}
					if (task.isDone) {
						task.pipe->asyncTextureEncoder->ClearTask(task.encoderTaskIndex);
					}
				}
			}
			catch (const Exception & e)
			{
				DEBUG_LOG("RTP: Exception during query task %d status. %s\n", task.encoderTaskIndex, e.message.c_str());
				task.isDone = true; task.isError = true; task.error = e.message;
			}
		}

		if (task.isDone) {
			g_doneTaskPtr = (g_doneTaskPtr + 1) % MAX_PENDING_TASK_COUNT;
		}
		else {
			//We can't "done" next task if current task is not done yet.
			break;
		}
	}

	DEBUG_LOG("Render thread polling finished\n");
}

UNITY_INTERFACE_EXPORT UnityRenderingEvent UNITY_INTERFACE_API NvPipe_GetRenderThreadPollFunc() {
	return NvPipe_RenderThreadPoll;
}

/*
Called in main thread, query status of encode task.
Only task error will be returned in **error. Other error goes to sharedError
*/
UNITY_INTERFACE_EXPORT void UNITY_INTERFACE_API NvPipe_EncodeTextureAsyncQuery(
	uint32_t taskIndex, bool* isDone, bool* isError, uint8_t** encodedData, uint64_t* encodeSize, const char** error) {

	try
	{
		//Check taskIndex is valid.
		if (taskIndex >= MAX_PENDING_TASK_COUNT || !CheckInsideQueueRange(g_pendingTaskPtr.load(), g_cleardTaskPtr.load(), taskIndex))
		{
			throw Exception("Task is not valid!");
		}

		if (CheckInsideQueueRange(g_pendingTaskPtr.load(), g_doneTaskPtr.load(), taskIndex)) {
			//Not done yet.
			*isDone = false;
			return;
		}

		if (CheckInsideQueueRange(g_doneTaskPtr.load(), g_cleardTaskPtr.load(), taskIndex)) {
			auto& task = mainThreadPendingTasks[taskIndex];
			//Small check
			if (task.isDone != true) {
				//This should never happen
				throw Exception("Fatal error.");
			}
			*isDone = true;
			*isError = task.isError;
			if (task.isError) {
				*error = task.error.c_str();
			}
			else {
				*encodeSize = task.encodedSize;
				*encodedData = task.resultBuffer;
			}
			return;
		}

		throw Exception("Unknown error....");
	}
	catch (const Exception & e)
	{
		sharedError = e.message;
	}

}

/*Called in main thread, to notify that a task could be cleared.*/
UNITY_INTERFACE_EXPORT void UNITY_INTERFACE_API NvPipe_EncodeTextureAsyncClearTask(uint32_t taskIndex) {
	try
	{
		if (g_cleardTaskPtr != taskIndex) {
			throw Exception("Only next uncleared task could be cleared");
		}
		if (taskIndex == g_doneTaskPtr) {
			throw Exception("The task is still being executed and can't be cleared");
		}

		DEBUG_LOG("RTP: %d is cleared \n", g_cleardTaskPtr.load());
		mainThreadPendingTasks[taskIndex] = MainThreadPendingTask();	//Clear it to empty object, to release references to pipe or buffer.

		g_cleardTaskPtr = (g_cleardTaskPtr + 1) % MAX_PENDING_TASK_COUNT;
	}
	catch (Exception & e)
	{
		sharedError = e.getErrorString();
		return;
	}
}

#endif

#endif

#ifdef NVPIPE_WITH_DECODER

UNITY_INTERFACE_EXPORT uint32_t UNITY_INTERFACE_API NvPipe_CreateDecoder(NvPipe_Format format, NvPipe_Codec codec, uint32_t width, uint32_t height)
{
	auto instance = std::make_shared<Instance>();

	try
	{
		instance->decoder = std::unique_ptr<Decoder>(new Decoder(format, codec, width, height));
		return InsertNewPipe(instance);
	}
	catch (Exception & e)
	{
		sharedError = e.getErrorString();
		return 0;
	}

	return 0;
}

UNITY_INTERFACE_EXPORT uint64_t UNITY_INTERFACE_API NvPipe_Decode(uint32_t nvp, const uint8_t* src, uint64_t srcSize, void* dst, uint32_t width, uint32_t height)
{
	auto instance = GetPipe(nvp);
	if (instance == nullptr)
		return 0;
	if (!instance->decoder)
	{
		instance->error = "Invalid NvPipe decoder.";
		return 0;
	}

	try
	{
		return instance->decoder->decode(src, srcSize, dst, width, height);
	}
	catch (Exception & e)
	{
		instance->error = e.getErrorString();
		return 0;
	}
}

#ifdef NVPIPE_WITH_OPENGL

UNITY_INTERFACE_EXPORT uint32_t UNITY_INTERFACE_API NvPipe_DecodeTexture(uint32_t nvp, const uint8_t* src, uint32_t srcSize, uint32_t texture, uint32_t target, uint32_t width, uint32_t height)
{
	auto instance = GetPipe(nvp);
	if (instance == nullptr)
		return 0;
	if (!instance->decoder)
	{
		instance->error = "Invalid NvPipe decoder.";
		return 0;
	}

	try
	{
		return instance->decoder->decodeTexture(src, srcSize, texture, target, width, height);
	}
	catch (Exception & e)
	{
		instance->error = e.getErrorString();
		return 0;
	}
}

#endif

#endif

UNITY_INTERFACE_EXPORT void UNITY_INTERFACE_API NvPipe_Destroy(uint32_t pipe)
{
	DeletePipe(pipe);
}

UNITY_INTERFACE_EXPORT const char* UNITY_INTERFACE_API NvPipe_GetError(uint32_t pipe)
{
	auto p = GetPipe(pipe);
	if (p == nullptr)
		return sharedError.c_str();

	return p->error.c_str();
}

UNITY_INTERFACE_EXPORT void UNITY_INTERFACE_API NvPipe_ClearError(uint32_t pipe)
{
	auto p = GetPipe(pipe);
	if (p == nullptr) {
		sharedError = "";
		return;
	}

	p->error = "";
}